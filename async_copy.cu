#include "hip/hip_runtime.h" 


#include <stdio.h>
 
__device__ int devData;
__host__ __device__ int run_on_cpu_or_gpu() {
	return 1;
}
 
__global__ void run_on_gpu() {
	printf("run_on_cpu_or_gpu GPU: %d\n", run_on_cpu_or_gpu());
}
 
int main() {
	int val = run_on_cpu_or_gpu();
	hipMemcpyToSymbol(HIP_SYMBOL(devData), &val, sizeof(int));
	printf("run_on_cpu_or_gpu CPU: %d\n", run_on_cpu_or_gpu());
	hipMemcpyFromSymbol(&val, HIP_SYMBOL(devData), sizeof(int));
	run_on_gpu<<<1, 1>>>();
	hipDeviceReset();
	return 0;
}

