#include "hip/hip_runtime.h" 


#include <stdio.h>
 

__host__ __device__ int run_on_cpu_or_gpu() {
	return 1;
}
 
__global__ void run_on_gpu() {
	printf("run_on_cpu_or_gpu GPU: %d\n", run_on_cpu_or_gpu());
}
 
int main() {

	printf("run_on_cpu_or_gpu CPU: %d\n", run_on_cpu_or_gpu());
	run_on_gpu<<<1, 1>>>();
	printf("will end\n");
	hipDeviceReset();
	return 0;
}